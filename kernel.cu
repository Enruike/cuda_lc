#include "hip/hip_runtime.h"
﻿#include "definitions.cuh"
#include<time.h>
//#include <bits/stdc++.h>
hipError_t cudaStatus;

int total_points = 0;
double dt = 0.0;
double dtime = 0.0;
int cycle = 0;
bool flag;
double S, S2, S0;

__device__ double d_trqq(double Qin[6]){
        double ans = 0.;
        ans = Qin[0] * Qin[0] + Qin[3] * Qin[3] + Qin[5] * Qin[5]\
                + 2 * (Qin[1] * Qin[1] + Qin[2] * Qin[2] + Qin[4] * Qin[4]);
        return ans;
}

/* __global__ void test_symbol(void){
	printf("L1_dev is %lf", L1_dev);
}
 */
__global__ void d_checktrace(double* d_Qold, unsigned int droplet){

	unsigned int indx = threadIdx.x + blockDim.x * blockIdx.x;

	if(indx < droplet){

		double tr = 0;
		double third =  1.0 / 3.0;
	
		tr = (d_Qold[indx * 6 + 0] + d_Qold[indx * 6 + 3] + d_Qold[indx * 6 + 5]) * third;

		if(tr > 1e-5) {
			
			printf("Correcting trace %lf for node %d!\n %f %f %f %f %f %f\n", tr, indx, d_Qold[indx * 6 + 0], d_Qold[indx * 6 + 1],\
				d_Qold[indx * 6 + 2], d_Qold[indx * 6 + 3], d_Qold[indx * 6 + 4], d_Qold[indx * 6 + 5]);

			d_Qold[indx * 6 + 0] -= tr;
			d_Qold[indx * 6 + 3] -= tr;
			d_Qold[indx * 6 + 5] -= tr;
			//printf("Non-tracelss.\n");			
		}

		double Qin[6] = { 0. };

		for(int i = 0; i < 6; i++){
			Qin[i] = d_Qold[indx * 6 + i];
		}

		if(d_trqq(Qin) > 1.){
	//              for(n = 0; n < 6; n ++){
	//                      Q[n] /= 1.3;
	//              }
			printf("Order parameter exceed 1. For node #%d\nQ info %f %f %f %f %f %f\nTrQQ:%lf\n", indx, d_Qold[indx * 6 + 0], d_Qold[indx * 6 + 1],\
				d_Qold[indx * 6 + 2], d_Qold[indx * 6 + 3], d_Qold[indx * 6 + 4], d_Qold[indx * 6 + 5], d_trqq(Qin));
		}
	}
}

int main() {

	//device variable callings
	double* d_Qold;
	unsigned char* d_Nvector_signal;
	unsigned char* d_Qtensor_signal;
	unsigned int* d_Nvector_index;
	unsigned int* d_Qtensor_index;

	unsigned char* d_bulktype;
	double* d_Qnew;
	int* d_neighbor;
	double* d_nu;
	double* d_Qo;

	flag = true;
	double time_taken;
	time_t start, end;

	time(&start);

	//Lectura de parámetros.
	//Si los parámetros son verdaderos continuará.
	if (!read_param()) {
		printf("No file param.in found!\n");
		//EXIT_SUCCESS();
		return 0;
		exit(1);
	}

	//continua con la funcion initial ubicada en initialization.cpp
	else {
		old_en = 1.;
		//S and U are the values for the inner LC. Also irx, iry and irz are the corresponding radii.
		S = 0.25 * (1.0 + 3.0 * sqrt(1.0 - 8.0 / (3.0 * U)));
		//S2 and U2 are the values for outer LC. Rx, Ry, Rz would be the radii of the whole droplet.
		S2 = 0.25 * (1.0 + 3.0 * sqrt(1.0 - 8.0 / (3.0 * U2)));
		dt = tmin;
		dtime = (tmax - tmin) / increment;
		total_points = Nx * Ny * Nz;

		printf("Value for S1 is: %lf\n", S);
		printf("Value for S2 is: %lf\n", S2);

		if (!initial()) {
			printf("Geometry couldn't be initialized!\n");
			//EXIT_SUCCESS();
			return 0;
		}

		else {
			printf("Geometry successfully initialized!\n");
		}
		//Freeing vectors used in geometry.
		

		unsigned int* h_Nvector_index;
		unsigned char* h_Nvector_signal;

		h_Nvector_index = (unsigned int*)malloc((surf + nsurf) * sizeof(unsigned int));
		h_Nvector_signal = (unsigned char*)malloc((surf + nsurf) * sizeof(unsigned char));

		unsigned int nb = 0;

		for (int i = 0; i < droplet; i++) {
			if ((signal[i] >= 2 && signal[i] <= 8) || (signal[i] == 12 || signal[i] == 13) || (signal[i] >= 20 && signal[i] <= 23)) {
				h_Nvector_index[nb] = i;				//We can find the Qtensor index of the Surface Vector point (nu_p or nu).
				h_Nvector_signal[nb] = signal[i];		//Type of point.
				nb++;
			}
		}

		if (nb != surf + nsurf) {
			printf("Error in transfer index and types for surface Qtensors!\n");
			printf("Count is %d, surf is %d & nsurf is %d!\n", nb, surf, nsurf);
			exit(1);
		}

		nb = 0;

		for (int i = 0; i < droplet; i++) {
			if ((signal[i] >= 2 && signal[i] <= 8) || (signal[i] == 12 || signal[i] == 13) || (signal[i] >= 20 && signal[i] <= 23)) {
				if (Qold[h_Nvector_index[nb]] != Qold[i]) {
					printf("Error in transfer from Qtensor to Surface Index Vector!\n");
					exit(1);
				}
				if (h_Nvector_signal[nb] != signal[i]) {
					printf("Error in transfer from Signal Vector to Surface Index Type Vector!\n");
					exit(1);
				}
				nb++;
			}
		}		

		unsigned int* h_Qtensor_index;
		unsigned char* h_Qtensor_signal;

		h_Qtensor_index = (unsigned int*)malloc(bulk * sizeof(unsigned int));
		h_Qtensor_signal = (unsigned char*)malloc(bulk * sizeof(unsigned char));

		nb = 0;

		for (int i = 0; i < droplet; i++) {
			if (signal[i] == 0 || signal[i] == 1) {
				h_Qtensor_index[nb] = i;
				h_Qtensor_signal[nb] = signal[i];
				nb++;
			}
		}

		if (nb != bulk) {
			printf("Error in transfer index and types for bulk Qtensors!\n");
			printf("Count is %d and bulk is %d!\n", nb, bulk);
			exit(1);
		}

		nb = 0;
		for(int i = 0; i < bulk;  i++) {
			if (signal[i] == 0 || signal[i] == 1) {
				if (Qold[h_Qtensor_index[nb]] != Qold[i]) {
					printf("Error in transfer from Qtensor to new Tensor Index Vector!\n");
					exit(1);
				}
				if (h_Qtensor_signal[nb] != signal[i]) {
					printf("Error in transfer from Signal Vector to new Tensor Index Type Vector!\n");
					exit(1);
				}
				nb++;
			}
		}
		nb = 0;

		//Remove signal vector in order to implement h_index_nu signal.

		// ********************************************** ----------------------- ****************************//
		//Allocating arrays in device memory. We add the error checker.
		cudaStatus = hipMalloc((void**)&d_Qold, sizeof(double) * droplet * 6);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}
		cudaStatus = hipMalloc((void**)&d_bulktype, sizeof(unsigned char) * droplet);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}
		if (infinite == 0 && degenerate == 0){
			//d_Q0 allocation
			cudaStatus = hipMalloc((void**)&d_Qo, sizeof(double) * (surf + nsurf) * 6);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMalloc failed!");
				return 0;
			}
		}
		// else{
		// 	cudaStatus = hipMalloc((void**)&d_Qo, 1 * surf * 6);
		// 	if (cudaStatus != hipSuccess) {
		// 		fprintf(stderr, "hipMalloc failed!");
		// 		return 0;
		// 	}
		// }

		//*************we now change de sizes of new signal vectors.********************//

		cudaStatus = hipMalloc((unsigned char**)&d_Qtensor_signal, sizeof(unsigned char) * bulk);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}

		cudaStatus = hipMalloc((void**)&d_Nvector_signal, sizeof(unsigned char) * (surf + nsurf));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}

		cudaStatus = hipMalloc((void**)&d_Qtensor_index, sizeof(unsigned int) * bulk);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}

		cudaStatus = hipMalloc((void**)&d_Nvector_index, sizeof(unsigned int) * (surf + nsurf));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}
		//****************************************************************************//

		//Neighboor must have sign for -1 value.
		cudaStatus = hipMalloc((void**)&d_neighbor, sizeof(int) * droplet * 6);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}
		cudaStatus = hipMalloc((void**)&d_nu, sizeof(double) * (surf + nsurf) * 3);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 0;
		}

		//We need h_bulktype to calculate energy in CPU. Don't free it.
		cudaStatus = hipMemcpy(d_bulktype, h_bulktype, droplet * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed bulktyoe!");
			return 0;
		}
		if (infinite == 0 && degenerate == 0){
			cudaStatus = hipMemcpy(d_Qo, Qo, (surf + nsurf) * 6 * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed Qo!");
				return 0;
			}
		}

		//Copy from host to device
		cudaStatus = hipMemcpy(d_Qold, Qold, droplet * 6 * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed Qtensor!");
			return 0;
		}

		////****************************///
		//New vectors signal
		cudaStatus = hipMemcpy(d_Nvector_signal, h_Nvector_signal, (surf + nsurf) * sizeof(unsigned char), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed Nvector Sign!");
			return 0;
		}

		cudaStatus = hipMemcpy(d_Qtensor_signal, h_Qtensor_signal, bulk * sizeof(unsigned char), hipMemcpyHostToDevice);
		if(cudaStatus != hipSuccess){
			fprintf(stderr, "hipMemcpy failed Qtensor signal!");
			return 0;
		}

		//New vectors for index
		cudaStatus = hipMemcpy(d_Nvector_index, h_Nvector_index, (surf + nsurf) * sizeof(unsigned int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed Nvector!");
			return 0;
		}
		cudaStatus = hipMemcpy(d_Qtensor_index, h_Qtensor_index, bulk * sizeof(unsigned int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed Qtensor index!");
			return 0;
		}

		//Freeing h_Qtensor_index, h_Nvector_index, h_Nvector_signal
		hipFree(h_Qtensor_index);
		hipFree(h_Nvector_index);
		hipFree(h_Nvector_signal);

		////****************************///

		cudaStatus = hipMemcpy(d_neighbor, neighbor, droplet * 6 * sizeof(signed int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed for neighboors!");
			return 0;
		}
		cudaStatus = hipMemcpy(d_nu, nu, (surf + nsurf) * 3 * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed surface vector!");
			return 0;
		}

		/*printQ << <1, 10 >> > (d_Nvector_signal);
		hipDeviceSynchronize();*/

		unsigned int threads_per_block = 512;
		unsigned int blk_thrds = 384;
		//size for surface
		unsigned int surfBlocks = rint((surf + nsurf) / threads_per_block) + 1;		

		//size for bulk
		unsigned int bulkBlocks = rint(bulk / threads_per_block) + 1;

		printf("The number of Bulk Blocks is %d\n", bulkBlocks);

		printf("The number of Surf Blocks is %d\n", surfBlocks);

		unsigned int dropletBlocks = rint((bulk + surf + nsurf) / threads_per_block) + 1;

		printf("The number of Droplet Blocks is %d\n\n", dropletBlocks);

		//__device__ double devThird;
		//double third = 1. / 3.;
		//hipMemcpyToSymbol(HIP_SYMBOL(devThird), &third, sizeof(double));
		//__host__ __device__ __constant__ double d_idx;
		//hipMemcpyToSymbol(HIP_SYMBOL(d_idx), &idx, sizeof(double));

		/* Copy symbols */
	/* 	hipMemcpyToSymbol(HIP_SYMBOL(chiral_dev), &chiral, sizeof(int));
		
		printf("host L1 value %lf", L1);

		cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(L1_dev), &L1, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "failed Memcpy Symbol!");
			return 0;
		}

		test_symbol<<<1,16>>>();

		hipMemcpyToSymbol(HIP_SYMBOL(L2_dev), &L2, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(L3_dev), &L3, sizeof(float));
		hipMemcpyToSymbol(HIP_SYMBOL(L4_dev), &L4, sizeof(float)); */

		//Progress bar
		const char *shade = "\u2592";
    	const char *shade2 = "\u2588";

		if(DoubleU && geo == 4){
			S0 = S2;
		}
		else{
			S0 = S;
		}
		
		if(stopat != 0){

			printf("Total Progress\n");
			printf("[");

			for(int i = 0; i < 50; i++){
				printf(" ");
			}
			
			printf("] 0.00%\n");
		}
	
		while (flag) {

			printf("\t\t ~Computing Energy~ \n");
			free_energy();
			
			if(fabs(dE) < accuracy || (stopat != 0 && cycle == stopat)){
				printf("Error in the trace of q; cycle : %d.\n", cycle);
				flag = false;
				break;
			}

			printf("\t\t ~Checking Trace~ \n");
			printf("\033[1;31m");
			d_checktrace<<<dropletBlocks, threads_per_block>>>(d_Qold, droplet);
			hipDeviceSynchronize();
			printf("\033[0m");

			if((cycle % save_every) == 0){
				printf("\x1b[32m");
				printf("\t\t ~Saving Data~ \n");
				printf("\033[0m");
				output();
			}

			printf("\t\t ~Relaxing~ \n");

			printf("\033[1;33m");
			//printf("\t");
			for(int i = 0; i < 50; i++){
				//std::cout << "\x2592";
				printf(shade);
			}

			printf("\r");
			//printf("\t");
			for (int i = 0; i < check_every; i++) {

				if((cycle%(check_every/50)==0)){
					printf(shade2);
				}

				/* test_symbol<<<1,32>>>();
				hipDeviceSynchronize(); */

				relax_bulk<<<bulkBlocks, threads_per_block>>>(d_Qold, d_bulktype, d_neighbor, d_Qtensor_index, d_Qtensor_signal,
					U, U2, chiral, qch,L1, L2, bulk, idx, idy, idz, iddx, iddy, iddz, dt);
				hipDeviceSynchronize(); 

				relax_surf<<<surfBlocks, threads_per_block>>>(d_Qold, d_neighbor, d_Nvector_index, d_Nvector_signal, d_Qo, chiral, qch, L1, L2, L3, L4,
					tiltAngle, (surf + nsurf), degenerate, infinite, W, Wp, d_nu, idx, idy, idz, dt, S0);

				hipDeviceSynchronize();
 
				if(dt < tmax){
					dt += dtime;
					if(dt >= tmax){
						dt = tmax;
					}
				}
				cycle++;
				//

			}
			printf("\033[0m\n");
			printf("\n");

			if(stopat != 0){

				double percentage = (double)cycle / (double)stopat * 100.;

				printf("Total Progress\n");
				printf("[");
				for(int i = 0; i < rint(percentage / 2); i++){
					printf("#");
				}
				for(int i = 0; i < rint(100 / 2) - rint(percentage / 2); i++){
					printf(" ");
				}
				
				printf("] %.2lf%\n", percentage);
			}

			//system("clear");
			//printf("\n\t\t\t ~Done~ \n\n");

			// flag=false;
			//printf("      <<=== ~Copying Q-Tensor back to Host Memory~ ===>>>\n");
			cudaStatus = hipMemcpy(Qold, d_Qold, droplet * 6 * sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!\n");
				return 0;
				break;
			}
		}
	
		

		output();
		
		time(&end);

		time_taken = (double)(end - start);

		FILE* energy;

		energy = fopen("energy.out", "a");
		if(time_taken < 60){
			fprintf(energy, "\nTime used:	%lf min.\n", time_taken);
			printf("\nTime used:	%lf min.\n", time_taken);
		}
		else{
			fprintf(energy, "\nTime used:	%lf h.\n", time_taken / 60. / 60.);
			printf("\nTime used:	%lf h.\n", time_taken / 60. / 60.);
		}
		fclose(energy);	

	//free device variables
		hipFree(d_Qold);
		if(infinite == 0 && degenerate == 0){
			hipFree(d_Qo);
		}
		
		hipFree(d_bulktype);
		hipFree(d_neighbor);
		hipFree(d_nu);
		hipFree(d_Qtensor_index);
		hipFree(d_Nvector_index);
		hipFree(d_Nvector_signal);

		//free host variables
		free(signal);
		if(infinite == 0 && degenerate == 0){
			free(Qo);
		}
		free(nu);
		free(Qold);
		free(neighbor);
		free(h_bulktype);
		free(drop);
		free(boundary);
		return true;
	}

}
